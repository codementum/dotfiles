/*
 * =================================================
 *
 *  Filename: main.cu
 *  Description: main file for timehisto project
 *
 *  Version:    1.0
 *  Created:    Wed Apr 27 19:48:09 EDT 2011
 *  Revision:   none
 *  Compiler:   nvcc
 *  
 *  Author: Lane Harrison, Ltharri1@uncc.edu 
 *
 * =================================================
 */

// Includes
#include <stdio.h>
#include <hip/hip_runtime.h>

// Global Variables

// Kernel Functions

__global__ void add( int a, int b, int *c ){
    *c = a + b;
}

// Regular Functions
int main(void) {
    int c;
    int *dev_c;

    hipMalloc( (void**)&dev_c, sizeof(int) );

    add<<<1, 1>>> ( 2, 7, dev_c );

    hipMemcpy( &c,
                dev_c,
                sizeof(int),
                hipMemcpyDeviceToHost );
    printf( "2 + 7 = %d\n", c );
    hipFree( dev_c );

    return 0;
}


